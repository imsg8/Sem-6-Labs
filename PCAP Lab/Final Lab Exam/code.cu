/*
 * Problem: Build a String from Two Matrices on GPU with Prefix Array
 *
 * In this problem, you will generate an output string by processing two matrices,
 * one containing integer counts and the other containing characters, using GPU
 * parallelism. The matrices share the same dimensions (m × n) and are flattened
 * into 1D arrays of length N = m * n. On the host, you compute for each element
 * how many times its corresponding character should appear—doubling the count
 * if the integer is prime or using it directly otherwise—and then build a prefix‐sum
 * array that records the starting index for each character’s block in the final string.
 * This prefix array and the character array are transferred to the device, where a
 * scatter kernel uses each thread to write its character into the correct segment
 * of a pre‐allocated output buffer. Once the GPU completes this parallel scatter,
 * the assembled buffer is copied back to the host, null‑terminated, and printed.
 * This design avoids write conflicts, balances work across threads, and scales
 * efficiently for large matrices (up to 1000×1000).
 *
 * Host‐Device Workflow:
 *   1. Flatten A[m][n] → A_flat[N], C[m][n] → C_flat[N].
 *   2. On host, compute lengths[k] = isPrime(A_flat[k]) ? 2*A_flat[k] : A_flat[k].
 *   3. Build prefix‑sum P of size N+1: P[0]=0; for k in [0..N-1], P[k+1]=P[k]+lengths[k].
 *   4. Copy C_flat and P to device memory.
 *   5. Launch scatterChars kernel to fill device output buffer.
 *   6. Copy result back to host, append '\0', and print the string.
 *
 * Function Signature:
 *   // Builds the string on the GPU and prints it on the host.
 *   void buildStringGPU(int m, int n, int A[m][n], char C[m][n]);
 *
 */


#include <stdio.h>
#include "hip/hip_runtime.h"


__device__ __host__ int isprime(int n) {
    if (n <= 1) return 0;
    for (int i = 2; i <= n / 2; ++i) {
        if (n % i == 0) return 0;
    }
    return 1;
}

__global__ void kernel(int *array, char *chararray, int *prefix, char *string, int m, int n) {
    int tid = threadIdx.x;
    if (tid >= m * n) return;

    int sid  = prefix[tid];             
    int temp = array[tid];              
    int nott = isprime(temp) ? 2*temp : temp; 

    for (int k = 0; k < nott; ++k) {
        string[sid + k] = chararray[tid];
    }
}

int main() {
    int m, n;
    printf("Enter m and n: ");
    scanf("%d %d", &m, &n);
    int size = m * n;

    printf("Enter int array: ");
    int array[size];
    for (int i = 0; i < size; ++i) {
        scanf("%d", &array[i]);
    }

    printf("Enter char array: ");
    char chararray[size];
    for (int i = 0; i < size; ++i) {
        scanf(" %c", &chararray[i]);
    }

    int prefix[size];
    int oldval = 0;
    for (int i = 0; i < size; ++i) {
        prefix[i] = oldval;
        int temp = array[i];
        oldval += isprime(temp) ? temp * 2 : temp;
    }

    int  *d_array, *d_prefix;
    char *d_chararray, *d_str;
    hipMalloc(&d_array,     sizeof(int)  * size);
    hipMalloc(&d_prefix,    sizeof(int)  * size);
    hipMalloc(&d_chararray, sizeof(char) * size);
    hipMalloc(&d_str,       sizeof(char) * oldval);

    hipMemcpy(d_array,     array,      sizeof(int)  * size, hipMemcpyHostToDevice);
    hipMemcpy(d_prefix,    prefix,     sizeof(int)  * size, hipMemcpyHostToDevice);
    hipMemcpy(d_chararray, chararray,  sizeof(char) * size, hipMemcpyHostToDevice);

    kernel<<<1, size>>>(d_array, d_chararray, d_prefix, d_str, m, n);
    hipDeviceSynchronize();

    char string[size * size];  
    hipMemcpy(string, d_str, sizeof(char) * oldval, hipMemcpyDeviceToHost);
    string[oldval] = '\0';

    printf("%s\n", string);

    hipFree(d_array);
    hipFree(d_prefix);
    hipFree(d_chararray);
    hipFree(d_str);

    return 0;
}
