// Credits: Tejinder Pokuri

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(int*da,int*db,int*dc){
    int index=threadIdx.x;
    dc[index]=da[index]+db[index];
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    int a[n],b[n],c[n];
    int *da,*db,*dc;

    hipMalloc((void **)&da,n*sizeof(int));
    hipMalloc((void **)&db,n*sizeof(int));
    hipMalloc((void **)&dc,n*sizeof(int));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        scanf("%d",&a[i]);

    printf("Enter vector two : ");
    for(int i=0;i<n;i++)
        scanf("%d",&b[i]);
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(1,1,1);
    dim3 blk(n,1,1);

    add_vec<<<grid,blk>>>(da,db,dc);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%d\t",c[i]);
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
